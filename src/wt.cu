#include "hip/hip_runtime.h"
#include "wt.h"
#include "common.h"

#ifdef SEPARATE_COMPILATION
// Required for separate compilation (see Makefile)
#ifndef CONSTMEM_FILTERS_S
#define CONSTMEM_FILTERS_S
__constant__ DTYPE c_kern_L[MAX_FILTER_WIDTH];
__constant__ DTYPE c_kern_H[MAX_FILTER_WIDTH];
__constant__ DTYPE c_kern_IL[MAX_FILTER_WIDTH];
__constant__ DTYPE c_kern_IH[MAX_FILTER_WIDTH];
#endif
#endif


/// Compute the low-pass and high-pass filters for separable convolutions.
/// wname: name of the filter ("haar", "db3", "sym4", ...)
/// Returns : the filter width "hlen" if success ; a negative value otherwise.
int w_compute_filters_separable(const char* wname) {
    int hlen = 0;
    DTYPE* f1_l, *f1_h, *f1_il, *f1_ih;

    // Browse available filters (see filters.h)
    int i;
    for (i = 0; i < 72; i++) {
        if (!strcasecmp(wname, all_filters[i].wname)) {
            hlen = all_filters[i].hlen;
            f1_l = all_filters[i].f_l;
            f1_h = all_filters[i].f_h;
            f1_il = all_filters[i].i_l;
            f1_ih = all_filters[i].i_h;
            break;
        }
    }
    if (hlen == 0) {
        printf("ERROR: w_compute_filters(): unknown filter %s\n", wname);
        return -2;
    }

    // Copy the filters to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_L), f1_l, hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_H), f1_h, hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_IL), f1_il, hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_IH), f1_ih, hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice);

    return hlen;
}

/// projection of the current reconstrucion (2D)
/// Must be lanched with block size (Nc, Nr) 
__global__ void kern_img_projc(DTYPE* d_image, DTYPE* Omega, DTYPE* y, int Nr, int Nc){
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;

    DTYPE val = 0.0f;
    if(gidy < Nr && gidx < Nc){
        val = d_image[gidy*Nc + gidx];
        d_image[gidy*Nc + gidx] = Omega[gidy*Nc + gidx]*val + (1.0f-Omega[gidy*Nc + gidx])*y[gidy*Nc + gidx];
    }    
}


void img_projc(DTYPE* d_image, DTYPE* Omega, DTYPE* y, int Nr, int Nc){
    int tpb = 32; // Threads per block
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    dim3 n_blocks;
    n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    kern_img_projc<<<n_blocks, n_threads_per_block>>>(d_image, Omega, y, Nr, Nc);
}

/// ----------------------------------------------------------------------------
/// --------------------- Forward Undecimated DWT ------------------------------
/// ----------------------------------------------------------------------------



// must be run with grid size = (Nc, Nr)  where Nr = numrows of input image
// Pass 1 : Input image ==> horizontal convolution with L, H  ==> (tmp_a1, tmp_a2)
__global__ void w_kern_forward_swt_pass1(DTYPE* img, DTYPE* tmp_a1, DTYPE* tmp_a2, int Nr, int Nc, int hlen, int level) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidy < Nr && gidx < Nc) {

        int factor = 1 << (level - 1);
        int c, hL, hR;
        if (hlen & 1) { // odd kernel size
            c = hlen/2;
            hL = c;
            hR = c;
        }
        else { // even kernel size : center is shifted to the left
            c = hlen/2 - 1;
            hL = c;
            hR = c+1;
        }

        c *= factor;
        int jx1 = c - gidx;
        int jx2 = Nc - 1 - gidx + c;
        DTYPE res_tmp_a1 = 0, res_tmp_a2 = 0;
        DTYPE img_val;

        // Convolution with periodic boundaries extension.
        // The filters are 2-upsampled at each level : [h0, h1, h2, h3] --> [h0, 0, h1, 0, h2, 0, h3, 0]
       for (int jx = 0; jx <= hR+hL; jx++) {
            int idx_x = gidx + jx*factor - c;
            if (factor*jx < jx1) idx_x += Nc;
            if (factor*jx > jx2) idx_x -= Nc;

            img_val = img[(gidy)*Nc + idx_x];
            res_tmp_a1 += img_val * c_kern_L[hlen-1 - jx];
            res_tmp_a2 += img_val * c_kern_H[hlen-1 - jx];
        }

        tmp_a1[gidy* Nc + gidx] = res_tmp_a1;
        tmp_a2[gidy* Nc + gidx] = res_tmp_a2;
    }
}

// must be run with grid size = (Nc, Nr)  where Nr = numrows of input image
// Pass 2 : (tmp_a1, tmp_a2) ==>  Vertic convolution on tmp_a1 and tmp_a2 with  L, H  ==> (a, h, v, d)
__global__ void w_kern_forward_swt_pass2(DTYPE* tmp_a1, DTYPE* tmp_a2, DTYPE* c_a, DTYPE* c_h, DTYPE* c_v, DTYPE* c_d, int Nr, int Nc, int hlen, int level) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidy < Nr && gidx < Nc) {

        int factor = 1 << (level - 1);
        int c, hL, hR;
        if (hlen & 1) { // odd kernel size
            c = hlen/2;
            hL = c;
            hR = c;
        }
        else { // even kernel size : center is shifted to the left
            c = hlen/2 - 1;
            hL = c;
            hR = c+1;
        }

        c *= factor;
        int jy1 = c - gidy;
        int jy2 = Nr - 1 - gidy + c;
        DTYPE res_a = 0, res_h = 0, res_v = 0, res_d = 0;

        // Convolution with periodic boundaries extension.
        // The filters are 2-upsampled at each level : [h0, h1, h2, h3] --> [h0, 0, h1, 0, h2, 0, h3, 0]
        for (int jy = 0; jy <= hR+hL; jy++) {
            int idx_y = gidy + factor*jy - c;
            if (factor*jy < jy1) idx_y += Nr;
            if (factor*jy > jy2) idx_y -= Nr;

            res_a += tmp_a1[idx_y*Nc + gidx] * c_kern_L[hlen-1 - jy];
            res_h += tmp_a1[idx_y*Nc + gidx] * c_kern_H[hlen-1 - jy];
            res_v += tmp_a2[idx_y*Nc + gidx] * c_kern_L[hlen-1 - jy];
            res_d += tmp_a2[idx_y*Nc + gidx] * c_kern_H[hlen-1 - jy];
        }

        c_a[gidy* Nc + gidx] = res_a;
        c_h[gidy* Nc + gidx] = res_h;
        c_v[gidy* Nc + gidx] = res_v;
        c_d[gidy* Nc + gidx] = res_d;
    }
}


int w_forward_swt_separable(DTYPE* d_image, DTYPE** d_coeffs, DTYPE* d_tmp, w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, levels = winfos.nlevels, hlen = winfos.hlen;

    DTYPE* d_tmp1 = d_tmp;
    DTYPE* d_tmp2 = d_tmp + Nr*Nc;

    int tpb = 32; // TODO : tune for max perfs.
    dim3 n_blocks_1 = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    dim3 n_blocks_2 = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    // First level
    w_kern_forward_swt_pass1<<<n_blocks_1, n_threads_per_block>>>(d_image, d_tmp1, d_tmp2, Nr, Nc, hlen, 1);
    w_kern_forward_swt_pass2<<<n_blocks_2, n_threads_per_block>>>(d_tmp1, d_tmp2, d_coeffs[0], d_coeffs[1], d_coeffs[2], d_coeffs[3], Nr, Nc, hlen, 1);
    // Other levels
    for (int i=1; i < levels; i++) {
        w_kern_forward_swt_pass1<<<n_blocks_1, n_threads_per_block>>>(d_coeffs[0], d_tmp1, d_tmp2, Nr, Nc, hlen, i+1);
        w_kern_forward_swt_pass2<<<n_blocks_2, n_threads_per_block>>>(d_tmp1, d_tmp2, d_coeffs[0], d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], Nr, Nc, hlen, i+1);
    }
    return 0;
}


/// ----------------------------------------------------------------------------
/// --------------------- Inverse Undecimated DWT ------------------------------
/// ----------------------------------------------------------------------------


// must be run with grid size = (Nc, Nr) ; Nr = numrows of input coefficients
// pass 1 : (a, h, v, d)  ==> Vertical convol with IL, IH  ==> (tmp1, tmp2)
__global__ void w_kern_inverse_swt_pass1(DTYPE* c_a, DTYPE* c_h, DTYPE* c_v, DTYPE* c_d, DTYPE* tmp1, DTYPE* tmp2, int Nr, int Nc, int hlen, int level) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidy < Nr && gidx < Nc) { // vertic oversampling : Input (Nr, Nc) => Output (Nr*2, Nc)
        int c, hL, hR;
        int factor = 1 << (level - 1);
        if (hlen & 1) { // odd half-kernel size
            c = hlen/2;
            hL = c;
            hR = c;
        }
        else { // even half-kernel size : center is shifted to the RIGHT for reconstruction.
            c = hlen/2 - 0;
            hL = c;
            hR = c-1;
        }
        c *= factor;
        int jy1 = c - gidy;
        int jy2 = Nr - 1 - gidy + c;
        int offset_y = 1-(gidy & 1);
        offset_y = 0;

        DTYPE res_a = 0, res_h = 0, res_v = 0, res_d = 0;
        for (int jy = 0; jy <= hR+hL; jy++) {
            int idx_y = gidy - c + factor*jy;
            if (factor*jy < jy1) idx_y += Nr;
            if (factor*jy > jy2) idx_y -= Nr;

            res_a += c_a[idx_y*Nc + gidx] * c_kern_IL[hlen-1 - (jy + offset_y)]/2;
            res_h += c_h[idx_y*Nc + gidx] * c_kern_IH[hlen-1 - (jy + offset_y)]/2;
            res_v += c_v[idx_y*Nc + gidx] * c_kern_IL[hlen-1 - (jy + offset_y)]/2;
            res_d += c_d[idx_y*Nc + gidx] * c_kern_IH[hlen-1 - (jy + offset_y)]/2;
        }
        tmp1[gidy * Nc + gidx] = res_a + res_h;
        tmp2[gidy * Nc + gidx] = res_v + res_d;
    }
}

// must be run with grid size = (Nr, Nc) ; Nc = numcols of input coeffs.
// pass 2 : (tmp1, tmp2)  ==> Horiz convol with IL, IH  ==> I
__global__ void w_kern_inverse_swt_pass2(DTYPE* tmp1, DTYPE* tmp2, DTYPE* img, int Nr, int Nc, int hlen, int level) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidy < Nr && gidx < Nc) { // horiz oversampling : Input (Nr*2, Nc) => Output (Nr*2, Nc*2)
        int c, hL, hR;
        int factor = 1 << (level - 1);
        if (hlen & 1) { // odd half-kernel size
            c = hlen/2;
            hL = c;
            hR = c;
        }
        else { // even half-kernel size : center is shifted to the RIGHT for reconstruction.
            c = hlen/2 - 0;
            hL = c;
            hR = c-1;
        }
        c *= factor;
        int jx1 = c - gidx;
        int jx2 = Nc - 1 - gidx + c;
        int offset_x = 1-(gidx & 1);
        offset_x = 0;

        DTYPE res_1 = 0, res_2 = 0;
        for (int jx = 0; jx <= hR+hL; jx++) {
            int idx_x = gidx - c + factor*jx;
            if (factor*jx < jx1) idx_x += Nc;
            if (factor*jx > jx2) idx_x -= Nc;

            res_1 += tmp1[gidy*Nc + idx_x] * c_kern_IL[hlen-1 - (jx + offset_x)]/2;
            res_2 += tmp2[gidy*Nc + idx_x] * c_kern_IH[hlen-1 - (jx + offset_x)]/2;
        }
        img[gidy * Nc + gidx] = res_1 + res_2;
    }
}


int w_inverse_swt_separable(DTYPE* d_image, DTYPE** d_coeffs, DTYPE* d_tmp, w_info winfos) {

    int Nr = winfos.Nr, Nc = winfos.Nc, levels = winfos.nlevels, hlen = winfos.hlen;
    DTYPE* d_tmp1 = d_tmp;
    DTYPE* d_tmp2 = d_tmp + Nr*Nc;

    int tpb = 32; // TODO : tune for max perfs.
    dim3 n_blocks_1 = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    dim3 n_blocks_2 = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);

    for (int i = levels-1; i >= 1; i--) {
        w_kern_inverse_swt_pass1<<<n_blocks_1, n_threads_per_block>>>(d_coeffs[0], d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], d_tmp1, d_tmp2, Nr, Nc, hlen, i+1);
        w_kern_inverse_swt_pass2<<<n_blocks_2, n_threads_per_block>>>(d_tmp1, d_tmp2, d_coeffs[0], Nr, Nc, hlen, i+1);
    }
    // First scale
    w_kern_inverse_swt_pass1<<<n_blocks_1, n_threads_per_block>>>(d_coeffs[0], d_coeffs[1], d_coeffs[2], d_coeffs[3], d_tmp1, d_tmp2, Nr, Nc, hlen, 1);
    w_kern_inverse_swt_pass2<<<n_blocks_2, n_threads_per_block>>>(d_tmp1, d_tmp2, d_image, Nr, Nc, hlen, 1);

    return 0;
}


